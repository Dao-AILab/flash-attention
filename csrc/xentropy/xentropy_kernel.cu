#include "hip/hip_runtime.h"
// Adapted from https://github.com/NVIDIA/apex/blob/master/apex/contrib/csrc/xentropy/xentropy_kernel.cu
// TD [2022-09-17]: We make it work for bfloat16, and add an option to do the backward inplace (to save memory).
/**
 * From PyTorch:
 *
 * Copyright (c) 2016-     Facebook, Inc            (Adam Paszke)
 * Copyright (c) 2014-     Facebook, Inc            (Soumith Chintala)
 * Copyright (c) 2011-2014 Idiap Research Institute (Ronan Collobert)
 * Copyright (c) 2012-2014 Deepmind Technologies    (Koray Kavukcuoglu)
 * Copyright (c) 2011-2012 NEC Laboratories America (Koray Kavukcuoglu)
 * Copyright (c) 2011-2013 NYU                      (Clement Farabet)
 * Copyright (c) 2006-2010 NEC Laboratories America (Ronan Collobert, Leon Bottou, Iain Melvin, Jason Weston)
 * Copyright (c) 2006      Idiap Research Institute (Samy Bengio)
 * Copyright (c) 2001-2004 Idiap Research Institute (Ronan Collobert, Samy Bengio, Johnny Mariethoz)
 *
 * From Caffe2:
 *
 * Copyright (c) 2016-present, Facebook Inc. All rights reserved.
 *
 * All contributions by Facebook:
 * Copyright (c) 2016 Facebook Inc.
 *
 * All contributions by Google:
 * Copyright (c) 2015 Google Inc.
 * All rights reserved.
 *
 * All contributions by Yangqing Jia:
 * Copyright (c) 2015 Yangqing Jia
 * All rights reserved.
 *
 * All contributions from Caffe:
 * Copyright(c) 2013, 2014, 2015, the respective contributors
 * All rights reserved.
 *
 * All other contributions:
 * Copyright(c) 2015, 2016 the respective contributors
 * All rights reserved.
 *
 * Caffe2 uses a copyright model similar to Caffe: each contributor holds
 * copyright over their contributions to Caffe2. The project versioning records
 * all such contribution and copyright details. If a contributor wants to further
 * mark their specific copyright on a particular contribution, they should
 * indicate their copyright solely in the commit message of the change when it is
 * committed.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * 3. Neither the names of Facebook, Deepmind Technologies, NYU, NEC Laboratories America
 *    and IDIAP Research Institute nor the names of its contributors may be
 *    used to endorse or promote products derived from this software without
 *    specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/NumericLimits.cuh>

// https://github.com/NVIDIA/apex/blob/master/csrc/type_shim.h
// #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
#define DISPATCH_FLOAT_AND_HALF_AND_BF16(TYPE, LEVEL, NAME, ...) \
  switch(TYPE) \
  { \
    case at::ScalarType::Float: \
    { \
      using scalar_t_##LEVEL = float; \
      __VA_ARGS__; \
      break; \
    } \
    case at::ScalarType::Half: \
    { \
      using scalar_t_##LEVEL = at::Half; \
      __VA_ARGS__; \
      break; \
    } \
    case at::ScalarType::BFloat16: \
    { \
      using scalar_t_##LEVEL = at::BFloat16; \
      __VA_ARGS__; \
      break; \
    } \
    default: \
      AT_ERROR(#NAME, " not implemented for '", toString(TYPE), "'");  \
  }
// #else
// #define DISPATCH_FLOAT_AND_HALF_AND_BF16(TYPE, LEVEL, NAME, ...) \
//   switch(TYPE) \
//   { \
//     case at::ScalarType::Float: \
//     { \
//       using scalar_t_##LEVEL = float; \
//       __VA_ARGS__; \
//       break; \
//     } \
//     case at::ScalarType::Half: \
//     { \
//       using scalar_t_##LEVEL = at::Half; \
//       __VA_ARGS__; \
//       break; \
//     } \
//     default: \
//       AT_ERROR(#NAME, " not implemented for '", toString(TYPE), "'");  \
//   }
// #endif

#define ALIGN_BYTES 16

using Tensor = at::Tensor;
using TensorList = at::TensorList;
using ScalarType = at::ScalarType;
using at::acc_type;

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxForwardEpilogue {
  __device__ __forceinline__ LogSoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : logsum(max_input + std::log(sum)) {}

  __device__ __forceinline__ LogSoftMaxForwardEpilogue(AccumT max_log_sum_exp)
    : logsum(max_log_sum_exp) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(input - logsum);
  }

  const AccumT logsum;
};

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxBackwardEpilogue {
  __device__ __forceinline__ LogSoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - std::exp(static_cast<AccumT>(output)) * sum);
  }

  const AccumT sum;
};



const int max_threads = 1024;

inline dim3 SoftMax_getBlockSize(int ILP, uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size / ILP, static_cast<uint64_t>(max_threads));
  while (block_size < (max_block_size/2)) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = std::max(block_size, static_cast<uint64_t>(32));
  return dim3(block_size);
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};


////////////////////////////////////////////////////////////////////////////////
// Regular kernel (fast when dim_size is large; requires inner_size == 1)
////////////////////////////////////////////////////////////////////////////////


template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template<typename T, typename AccumT>
struct AddFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + std::exp(v - max_k);
  }

  const AccumT max_k;
};

template <template<typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT>& r,
            AccumT defaultVal)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  uint32_t mask = (((uint64_t)1) << (blockDim.x / 32)) - 1;
  if (threadIdx.x < 32) {
    int lane = threadIdx.x % 32;
    if (lane < blockDim.x / 32) {
#pragma unroll
      for (int i = 0; i < 32; ++i) {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }
      __syncwarp(mask);
      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / 32; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename> class Reduction1, template<typename> class Reduction2, typename AccumT>
__device__ __forceinline__ void
blockReduce(AccumT* smem,
            AccumT* reducVal1,
            AccumT val1,
            const Reduction1<AccumT>& r1,
            AccumT defaultVal1,
            AccumT* reducVal2,
            AccumT val2,
            const Reduction2<AccumT>& r2,
            AccumT defaultVal2)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val1;
  smem[blockDim.x + threadIdx.x] = val2;

  __syncthreads();

  AccumT warpVal1 = defaultVal1;
  AccumT warpVal2 = defaultVal2;

  // First warp will perform per-warp reductions for the remaining warps
  uint32_t mask = (((uint64_t)1) << (blockDim.x / 32)) - 1;
  if (threadIdx.x < 32) {
    int lane = threadIdx.x % 32;
    if (lane < blockDim.x / 32) {
#pragma unroll
      for (int i = 0; i < 32; ++i) {
        warpVal1 = r1(warpVal1, smem[lane * 32 + i]);
        warpVal2 = r2(warpVal2, smem[lane * 32 + i + blockDim.x]);
      }
      __syncwarp(mask);
      smem[lane] = warpVal1;
      smem[lane + blockDim.x] = warpVal2;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal1 = defaultVal1;
  AccumT blockVal2 = defaultVal2;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / 32; ++i) {
      blockVal1 = r1(blockVal1, smem[i]);
      blockVal2 = r2(blockVal2, smem[i + blockDim.x]);
    }
    smem[0] = blockVal1;
    smem[blockDim.x] = blockVal2;
  }

  // Sync and broadcast
  __syncthreads();
  *reducVal1 = smem[0];
  *reducVal2 = smem[blockDim.x];
  __syncthreads();
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT
ilpReduce(int shift,
          T* data,
          int size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LoadT;
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    data -= shift;
    size += shift;
    if(threadIdx.x >= shift){
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  int last = size % (ILP * blockDim.x);

  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];

    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }

  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);

  return threadVal;
}

template <template<typename, typename> class Reduction1, template<typename, typename> class Reduction2, int ILP, typename T, typename AccumT>
__device__ __forceinline__ void
ilpReduce(int shift,
          T* data,
          int size,
          AccumT* reducVal1,
          const Reduction1<T, AccumT>& r1,
          AccumT defaultVal1,
          AccumT* reducVal2,
          const Reduction2<T, AccumT>& r2,
          AccumT defaultVal2)
{
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LoadT;

  AccumT threadVal1 = defaultVal1;
  AccumT threadVal2 = defaultVal2;
  int offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    data -= shift;
    size += shift;
    if(threadIdx.x >= shift){
      threadVal1 = r1(threadVal1, data[offset]);
      threadVal2 = r2(threadVal2, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  int last = size % (ILP * blockDim.x);

  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];

    for (int j = 0; j < ILP; ++j) {
      threadVal1 = r1(threadVal1, v[j]);
      threadVal2 = r2(threadVal2, v[j]);
    }
  }

  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x) {
    threadVal1 = r1(threadVal1, data[offset]);
    threadVal2 = r2(threadVal2, data[offset]);
  }

  *reducVal1 = threadVal1;
  *reducVal2 = threadVal2;
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxXEntropyForward(
    accscalar_t *losses,
    outscalar_t *max_log_sum_exp,
    scalar_t *input,
    int64_t *labels,
    int64_t classes,
    const float smoothing,
    const int total_classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  //output += blockIdx.x * classes;
  const int shift = ((uint64_t)input) % ALIGN_BYTES / sizeof(scalar_t);

  int64_t label = labels[blockIdx.x];

  // find the max and sum
  accscalar_t threadMax, threadSum, max_k, sum_k;
  ilpReduce<MaxFloat, AddFloat, ILP, scalar_t, accscalar_t>(
    shift, input, classes,
    &threadMax, MaxFloat<scalar_t, accscalar_t>(),
    -at::numeric_limits<accscalar_t>::max(),
    &threadSum, AddFloat<scalar_t, accscalar_t>(),
    static_cast<accscalar_t>(0));

  blockReduce<Max, Add, accscalar_t>(
      sdata,
      &max_k, threadMax, Max<accscalar_t>(),
      -at::numeric_limits<accscalar_t>::max(),
      &sum_k, threadSum, Add<accscalar_t>(),
      static_cast<accscalar_t>(0));

  accscalar_t threadExp = ilpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  // calculate per element loss with label smoothing
  // reserve max + log_sum_exp for bprop
  if (threadIdx.x == 0) {
    accscalar_t lse = max_k + std::log(sumAll);
    accscalar_t log_prob = (label >= 0 && label < classes) ? epilogue(static_cast<accscalar_t>(input[label])) : 0.f;
    losses[blockIdx.x] = (lse - sum_k / total_classes) * smoothing - log_prob * (1 - smoothing);
    max_log_sum_exp[blockIdx.x] = lse;
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t>
__device__ __forceinline__ void
apply(scalar_t *gradInput,
      scalar_t *logits,
      outscalar_t *max_log_sum_exp,
      outscalar_t *gradOutput,
      int64_t *labels,
      const float smoothing,
      int classes,
      const int total_classes)
{
  accscalar_t smooth_positives = 1.0 - smoothing;
  accscalar_t smooth_negatives = smoothing / total_classes;
  accscalar_t tmpGradOutput = gradOutput[blockIdx.x];
  int64_t label = labels[blockIdx.x];
  accscalar_t coeff = max_log_sum_exp[blockIdx.x];

  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);

  for (; offset < classes - last; offset += blockDim.x * ILP) {
    accscalar_t tmpLogits[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmpLogits[j] = static_cast<accscalar_t>(logits[offset + j * blockDim.x]);
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
      gradInput[offset + j * blockDim.x] = tmpGradOutput * (
        std::exp(tmpLogits[j] - coeff) - static_cast<accscalar_t>(
          (offset + j * blockDim.x == label) ? 1 : 0) *
        smooth_positives - smooth_negatives);
  }

  for (; offset < classes; offset += blockDim.x)
    gradInput[offset] = tmpGradOutput * (std::exp(
        static_cast<accscalar_t>(logits[offset]) - coeff) -
        static_cast<accscalar_t>((offset == label) ? 1 : 0) *
        smooth_positives - smooth_negatives);
}


template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t>
__device__ __forceinline__ void
aligned_apply(int shift,
              scalar_t *gradInput,
              scalar_t *logits,
              outscalar_t *max_log_sum_exp,
              outscalar_t *gradOutput,
              int64_t *labels,
              const float smoothing,
              int classes,
              const int total_classes)
{
  accscalar_t smooth_positives = 1.0 - smoothing;
  accscalar_t smooth_negatives = smoothing / total_classes;
  accscalar_t tmpGradOutput = gradOutput[blockIdx.x];
  int64_t label = labels[blockIdx.x];
  accscalar_t coeff = max_log_sum_exp[blockIdx.x];

  int offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    logits -= shift;
    gradInput -= shift;
    classes += shift;
    if(threadIdx.x >= shift){
      gradInput[offset] = tmpGradOutput * (std::exp(
        static_cast<accscalar_t>(logits[offset]) - coeff) -
        static_cast<accscalar_t>(((offset - shift) == label) ? 1 : 0) *
        smooth_positives - smooth_negatives);
    }
    classes -= blockDim.x;
    gradInput += blockDim.x;
    logits += blockDim.x;
    shift -= blockDim.x;
  }

  int last = classes % (ILP * blockDim.x);

  typedef typename std::aligned_storage<ILP*sizeof(scalar_t), ILP*alignof(scalar_t)>::type LoadT;
  // input
  scalar_t v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);
  // output
  scalar_t r[ILP];
  LoadT* result = reinterpret_cast<LoadT*>(&r);

  for (; offset * ILP < (classes - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(logits)[offset];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      r[j] = tmpGradOutput * (std::exp(
          static_cast<accscalar_t>(v[j]) - coeff) -
          static_cast<accscalar_t>(((ILP * offset + j - shift) == label) ? 1 : 0) *
          smooth_positives - smooth_negatives);
    }
    reinterpret_cast<LoadT*>(gradInput)[offset] = *result;
  }

  offset = classes - last + threadIdx.x;
  for (; offset < classes; offset += blockDim.x)
    gradInput[offset] = tmpGradOutput * (std::exp(
        static_cast<accscalar_t>(logits[offset]) - coeff) -
        static_cast<accscalar_t>(((offset - shift) == label) ? 1 : 0) *
        smooth_positives - smooth_negatives);

}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxXEntropyBackward(
    scalar_t *gradInput,
    scalar_t *logits,
    outscalar_t *max_log_sum_exp,
    outscalar_t *gradOutput,
    int64_t *labels,
    const float smoothing,
    int classes,
    const int total_classes)
{
  gradInput += blockIdx.x * classes;
  logits += blockIdx.x * classes;

  // Do vectorized load/store when input/output have same alignment
  const int shift = ((uint64_t)logits) % ALIGN_BYTES / sizeof(scalar_t);
  const int shift_ = ((uint64_t)gradInput) % ALIGN_BYTES / sizeof(scalar_t);
  if (shift == shift_){
    aligned_apply<ILP, scalar_t, accscalar_t, outscalar_t>(shift, gradInput, logits, max_log_sum_exp, gradOutput, labels, smoothing, classes, total_classes <= 0 ? classes : total_classes);
  }
  else {
    apply<ILP, scalar_t, accscalar_t, outscalar_t>(gradInput, logits, max_log_sum_exp, gradOutput, labels, smoothing, classes, total_classes <= 0 ? classes : total_classes);
  }

}

template<template<typename, typename, typename> class Epilogue>
std::vector<Tensor> host_softmax_xentropy(
        const Tensor & input_,
        const Tensor & labels_,
        const float smoothing,
        const int total_classes) {
  // For tensor parallel cross entropy with smoothing, we want to pass in the total number
  // of classes so that smoothing can be applied correctly. If total_classes=-1, use the
  // last dimension of the input tensor.
  AT_ASSERTM(labels_.scalar_type() == ScalarType::Long,"Label type should be CUDA Long");

  // Otherwise the kernel will be launched from cuda:0 device
  // Cast to char to avoid compiler warning about narrowing
  at::cuda::CUDAGuard device_guard{(char)input_.get_device()};

  auto input = input_.contiguous();
  Tensor max_log_sum_exp = at::empty_like(labels_, input.options().dtype(ScalarType::Float));
  Tensor losses = at::empty_like(labels_, input_.options().dtype(ScalarType::Float));

  static_assert(std::is_same<acc_type<at::Half, true>, float>::value ||
    std::is_same<acc_type<at::Half, true>, double>::value,
    "accscalar_t for half should be float or double");
  AT_ASSERTM(input.dim() == 2, "Currently only 2 dim input supported");
  AT_ASSERTM(labels_.dim() == 1, "Labels should be 1 dimensional");
  AT_ASSERTM(input.size(0) == labels_.size(0), "Input and label should have same number of examples");
  AT_ASSERTM(input.numel() > 0, "Number of classes in input should not be 0");

  const int64_t dim = 1;
  int64_t outer_size = 1;
  int64_t dim_size = input.size(dim);
  int64_t inner_size = 1;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  for (int64_t i = 0; i < dim; ++i)
    outer_size *= input.size(i);
  for (int64_t i = dim + 1; i < input.dim(); ++i)
    inner_size *= input.size(i);
  // This kernel spawns a block per each element in the batch.
  // XXX: it assumes that inner_size == 1
  TORCH_CHECK(inner_size == 1, "Currently only inner size 1 supported");

  dim3 grid(outer_size);

  using namespace at;
  DISPATCH_FLOAT_AND_HALF_AND_BF16(input.scalar_type(), 0, "host_softmax_xentropy",
    using accscalar_t = at::acc_type<scalar_t_0, true>;
    const int ILP = sizeof(float4)/sizeof(scalar_t_0);
    dim3 block = SoftMax_getBlockSize(ILP, dim_size);
    cunn_SoftMaxXEntropyForward<ILP, scalar_t_0, accscalar_t, accscalar_t, Epilogue>
      <<<grid, block, 2 * block.x * sizeof(accscalar_t), stream>>>(
        losses.data_ptr<accscalar_t>(), max_log_sum_exp.data_ptr<accscalar_t>(),
        input.data_ptr<scalar_t_0>(), labels_.data_ptr<int64_t>(),
        dim_size, smoothing, total_classes <= 0 ? dim_size : total_classes
    );
  );

  C10_CUDA_CHECK(hipGetLastError());

  std::vector<at::Tensor> ret = {losses, max_log_sum_exp};
  return ret;
}

template<template<typename, typename, typename> class Epilogue>
Tensor host_softmax_xentropy_backward(
    const at::Tensor &grad_loss,
    at::Tensor &logits_,
    const at::Tensor &max_log_sum_exp,
    const at::Tensor &labels,
    const float smoothing,
    bool inplace,
    const int total_classes) {
  // Otherwise the kernel will be launched from cuda:0 device
  // Cast to char to avoid compiler warning about narrowing
  at::cuda::CUDAGuard device_guard{(char)grad_loss.get_device()};

  const int64_t dim = 1;
  Tensor gI = inplace ? logits_ : at::empty_like(logits_);
  if (grad_loss.numel() == 0) {
    return gI;
  }

  auto grad = grad_loss.contiguous();
  auto logits = logits_.contiguous();

  static_assert(std::is_same<acc_type<at::Half, true>, float>::value ||
    std::is_same<acc_type<at::Half, true>, double>::value,
    "accscalar_t for half should be float or double");
  if (grad.dim() == 0) grad = grad.view(1);

  AT_ASSERTM(logits_.dim() == 2, "Currently only 2 dim input supported");
  AT_ASSERTM(labels.dim() == 1, "Labels should be 1 dimensional");
  AT_ASSERTM(logits_.numel() > 0, "Number of classes in input should not be 0");
  AT_ASSERTM(logits_.size(0) == labels.size(0), "Input and label should have same number of examples");
  AT_ASSERTM(labels.size(0) == grad.size(0), "Label and loss should have same number of examples");

  int64_t outer_size = 1;
  int64_t dim_size = logits.size(dim);
  int64_t inner_size = 1;
  for (int64_t i = 0; i < dim; ++i)
    outer_size *= logits.size(i);
  for (int64_t i = dim + 1; i < logits.dim(); ++i)
    inner_size *= logits.size(i);
  // See descriptions of kernels above.
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  TORCH_CHECK(inner_size == 1, "Currently only inner size 1 supported");

  dim3 grid(outer_size);

  DISPATCH_FLOAT_AND_HALF_AND_BF16(gI.scalar_type(), 0, "host_softmax_xentropy_backward",
    using accscalar_t = acc_type<scalar_t_0, true>;
    const int ILP = sizeof(float4)/sizeof(scalar_t_0);
    dim3 block = SoftMax_getBlockSize(ILP, dim_size);
    cunn_SoftMaxXEntropyBackward<ILP, scalar_t_0, accscalar_t, accscalar_t, Epilogue>
      <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
        gI.data_ptr<scalar_t_0>(), logits.data_ptr<scalar_t_0>(),
        max_log_sum_exp.data_ptr<accscalar_t>(),
        grad.data_ptr<accscalar_t>(), labels.data_ptr<int64_t>(),
        smoothing, dim_size, total_classes
    );
  );

  C10_CUDA_CHECK(hipGetLastError());
  return gI;
}

std::vector<Tensor> softmax_xentropy_cuda(const Tensor &input, const Tensor &labels, const float smoothing, const int total_classes){
  return host_softmax_xentropy<LogSoftMaxForwardEpilogue>(input, labels, smoothing, total_classes);
}

at::Tensor softmax_xentropy_backward_cuda(
    const at::Tensor &grad_loss,
    at::Tensor &logits,
    const at::Tensor &max_log_sum_exp,
    const at::Tensor &labels,
    const float smoothing,
    const bool inplace,
    const int total_classes) {
  AT_ASSERTM((grad_loss.scalar_type() == ScalarType::Float), "expected grad types to be at::Float");
  return host_softmax_xentropy_backward<LogSoftMaxBackwardEpilogue>(grad_loss, logits, max_log_sum_exp, labels, smoothing, inplace, total_classes);
}
