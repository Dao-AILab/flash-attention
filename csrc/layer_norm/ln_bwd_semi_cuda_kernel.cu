#include "ln.h"
#include "ln_utils.cuh"
#include "ln_kernel_traits.h"
#include "ln_bwd_kernels.cuh"
#include "static_switch.h"

using namespace layer_norm;

template<
    typename weight_t,
    typename input_t,
    typename residual_t,
    typename output_t,
    typename compute_t,
    typename index_t,
    int HIDDEN_SIZE, 
    int CTAS_PER_ROW, 
    int WARPS_M, 
    int WARPS_N, 
    int BYTES_PER_LDG_MAIN,
    int BYTES_PER_LDG_FINAL
>
void launch_(LaunchParams<BwdParams> &launch_params, const bool configure_params, const bool prenorm){

    using Kernel_traits = Kernel_traits<weight_t,
                                        input_t,
                                        residual_t,
                                        output_t,
                                        compute_t,
                                        index_t,
                                        HIDDEN_SIZE,
                                        CTAS_PER_ROW,
                                        WARPS_M,
                                        WARPS_N,
                                        BYTES_PER_LDG_MAIN
                                        >;
    bool is_dropout = launch_params.params.dropout_keep_p < 1.f;
    bool has_residual = launch_params.params.dx1 != nullptr;
    bool has_rowscale = launch_params.params.rowscale != nullptr;
    BOOL_SWITCH(prenorm, PrenormConst, [&] {
        BOOL_SWITCH(is_dropout, IsDropoutConst, [&] {
            BOOL_SWITCH(has_residual, HasResidualConst, [&] {
                BOOL_SWITCH(has_rowscale, HasRowscaleConst, [&] {
                    auto kernel = &ln_bwd_kernel<Kernel_traits, PrenormConst, IsDropoutConst, HasResidualConst, HasRowscaleConst>;
                    if( configure_params ) {
                        int ctas_per_sm;
                        hipError_t status_ = hipOccupancyMaxActiveBlocksPerMultiprocessor(
                            &ctas_per_sm, kernel, Kernel_traits::THREADS_PER_CTA, Kernel_traits::SMEM_BYTES);
                        launch_params.params.ctas_per_col = launch_params.props->multiProcessorCount * ctas_per_sm / Kernel_traits::CTAS_PER_ROW;
                        launch_params.barrier_size = 0;
                        launch_params.workspace_bytes = 0;
                        if(Kernel_traits::CTAS_PER_ROW > 1) {
                            launch_params.barrier_size = 2 * launch_params.params.ctas_per_col;
                            launch_params.workspace_bytes = launch_params.params.ctas_per_col
                                                          * Kernel_traits::WARPS_M
                                                          * Kernel_traits::CTAS_PER_ROW
                                                          * sizeof(typename Kernel_traits::reduce_t)
                                                          * 2;
                        }
                        return;
                    }

                    if( Kernel_traits::SMEM_BYTES >= 48 * 1024 ) {
                        CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, Kernel_traits::SMEM_BYTES));
                    }
                    auto stream = launch_params.stream;
                    auto ctas_per_col = launch_params.params.ctas_per_col;

                    if( Kernel_traits::CTAS_PER_ROW == 1 ) {
                        kernel<<<ctas_per_col, Kernel_traits::THREADS_PER_CTA, Kernel_traits::SMEM_BYTES, stream>>>(launch_params.params);
                    } else {
                        dim3 grid(Kernel_traits::CTAS_PER_ROW * ctas_per_col);
                        dim3 block(Kernel_traits::THREADS_PER_CTA);
                        void *params_ = (void *)&launch_params.params;
                        hipLaunchCooperativeKernel((void *)kernel, grid, block, (void **)&params_, Kernel_traits::SMEM_BYTES, stream);
                    }

                    using Kernel_traits_f = layer_norm::Kernel_traits_finalize<HIDDEN_SIZE,
                                                                              weight_t,
                                                                              input_t,
                                                                              residual_t,
                                                                              output_t,
                                                                              compute_t,
                                                                              index_t,
                                                                              32 * 32,  // THREADS_PER_CTA
                                                                              BYTES_PER_LDG_FINAL>;

                    auto kernel_f = &layer_norm::ln_bwd_finalize_kernel<Kernel_traits_f>;
                    kernel_f<<<Kernel_traits_f::CTAS, Kernel_traits_f::THREADS_PER_CTA, 0, stream>>>(launch_params.params);
                });
            });
        });
    });
}

// Create backward launch function and register. Macro signature:
//  HIDDEN_SIZE, WTYPE, ITYPE, RTYPE, OTYPE, CTYPE, CTAS_PER_ROW, WARPS_M, WARPS_N, BYTES_PER_LDG, BYTES_PER_LDG_FINAL

REGISTER_BWD_LAUNCHER(  768, fp32, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp16, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp32, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp16, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp32, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp32, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, bf16, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp32, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, fp16, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER(  768, bf16, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_LAUNCHER( 1024, fp32, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp16, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp32, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp16, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp32, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp32, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, bf16, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp32, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, fp16, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1024, bf16, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_LAUNCHER( 1280, fp32, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp16, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp32, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp16, fp16, fp32, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp32, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp32, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, bf16, bf16, fp32, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp32, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, fp16, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_LAUNCHER( 1280, bf16, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_LAUNCHER( 1536, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 1536, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 1536, fp32, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, fp16, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, fp32, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, fp32, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, bf16, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, fp32, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, fp16, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 1536, bf16, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);

REGISTER_BWD_LAUNCHER( 1600, fp32, fp32, fp32, fp32, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp16, fp32, fp32, fp32, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp32, fp16, fp32, fp16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp16, fp16, fp32, fp16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp32, fp16, fp16, fp16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp32, bf16, fp32, bf16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, bf16, bf16, fp32, bf16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp32, bf16, bf16, bf16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, fp16, fp16, fp16, fp16, fp32, 1, 2, 1,  4, 4);
REGISTER_BWD_LAUNCHER( 1600, bf16, bf16, bf16, bf16, fp32, 1, 2, 1,  4, 4);

REGISTER_BWD_LAUNCHER( 2048, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp32, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp16, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp32, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp32, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, bf16, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp32, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, fp16, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2048, bf16, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

REGISTER_BWD_LAUNCHER( 2560, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2560, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 2560, fp32, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, fp16, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, fp32, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, fp32, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, bf16, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, fp32, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, fp16, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
REGISTER_BWD_LAUNCHER( 2560, bf16, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);

REGISTER_BWD_LAUNCHER( 3072, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp32, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp16, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp32, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp32, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, bf16, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp32, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, fp16, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 3072, bf16, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

REGISTER_BWD_LAUNCHER( 4096, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp32, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp16, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp32, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp32, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, bf16, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp32, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, fp16, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 4096, bf16, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

REGISTER_BWD_LAUNCHER( 5120, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp16, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp32, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp16, fp16, fp32, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp32, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp32, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, bf16, bf16, fp32, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp32, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, fp16, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_LAUNCHER( 5120, bf16, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

// TD [2022-04-22] Disable most of these to speed up compile time

// REGISTER_BWD_LAUNCHER( 1536, fp32, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER( 1536, fp32, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 1536, fp32, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 1536, fp32, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 1536, fp32, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 1536, fp16, fp16, fp16, fp16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 1536, bf16, bf16, bf16, bf16, fp32, 1, 1, 4,  8, 4);

// REGISTER_BWD_LAUNCHER( 2304, fp32, fp32, fp32, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 2304, fp16, fp16, fp16, fp32, 1, 1, 4,  4, 4);
// REGISTER_BWD_LAUNCHER( 2304, fp16, fp32, fp16, fp32, 1, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER( 2304, bf16, bf16, bf16, fp32, 1, 1, 4,  4, 4);
// REGISTER_BWD_LAUNCHER( 2304, bf16, fp32, bf16, fp32, 1, 1, 4,  8, 4);

// REGISTER_BWD_LAUNCHER( 3840, fp32, fp32, fp32, fp32, 1, 1, 4, 8, 4);
// REGISTER_BWD_LAUNCHER( 3840, fp16, fp16, fp16, fp32, 1, 1, 4, 4, 4);
// REGISTER_BWD_LAUNCHER( 3840, fp16, fp32, fp16, fp32, 1, 1, 4, 8, 4);
// REGISTER_BWD_LAUNCHER( 3840, bf16, bf16, bf16, fp32, 1, 1, 4, 4, 4);
// REGISTER_BWD_LAUNCHER( 3840, bf16, fp32, bf16, fp32, 1, 1, 4, 8, 4);

// REGISTER_BWD_LAUNCHER( 6144, fp32, fp32, fp32, fp32, 1, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER( 6144, fp16, fp16, fp16, fp32, 1, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER( 6144, fp16, fp32, fp16, fp32, 1, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER( 6144, bf16, bf16, bf16, fp32, 1, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER( 6144, bf16, fp32, bf16, fp32, 1, 1, 8, 16, 4);

// REGISTER_BWD_LAUNCHER( 8192, fp32, fp32, fp32, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER( 8192, fp16, fp16, fp16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER( 8192, fp16, fp32, fp16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER( 8192, bf16, bf16, bf16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER( 8192, bf16, fp32, bf16, fp32, 2, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(10240, fp32, fp32, fp32, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(10240, fp16, fp16, fp16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(10240, fp16, fp32, fp16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(10240, bf16, bf16, bf16, fp32, 2, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(10240, bf16, fp32, bf16, fp32, 2, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(12288, fp32, fp32, fp32, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12288, fp16, fp16, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12288, fp16, fp32, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12288, bf16, bf16, bf16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12288, bf16, fp32, bf16, fp32, 4, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(12800, fp32, fp32, fp32, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12800, fp16, fp16, fp16, fp32, 5, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(12800, fp16, fp32, fp16, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(12800, bf16, bf16, bf16, fp32, 5, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(12800, bf16, fp32, bf16, fp32, 5, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(15360, fp32, fp32, fp32, fp32, 4, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(15360, fp16, fp16, fp16, fp32, 4, 1, 4,  4, 4);
// REGISTER_BWD_LAUNCHER(15360, fp16, fp32, fp16, fp32, 4, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(15360, bf16, bf16, bf16, fp32, 4, 1, 4,  4, 4);
// REGISTER_BWD_LAUNCHER(15360, bf16, fp32, bf16, fp32, 4, 1, 4,  8, 4);

// REGISTER_BWD_LAUNCHER(16384, fp32, fp32, fp32, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(16384, fp16, fp16, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(16384, fp16, fp32, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(16384, bf16, bf16, bf16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(16384, bf16, fp32, bf16, fp32, 4, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(18432, fp32, fp32, fp32, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(18432, fp16, fp16, fp16, fp32, 4, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(18432, fp16, fp32, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(18432, bf16, bf16, bf16, fp32, 4, 1, 4,  8, 4);
// REGISTER_BWD_LAUNCHER(18432, bf16, fp32, bf16, fp32, 4, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(20480, fp32, fp32, fp32, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(20480, fp16, fp16, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(20480, fp16, fp32, fp16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(20480, bf16, bf16, bf16, fp32, 4, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(20480, bf16, fp32, bf16, fp32, 4, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(24576, fp32, fp32, fp32, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(24576, fp16, fp16, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(24576, fp16, fp32, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(24576, bf16, bf16, bf16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(24576, bf16, fp32, bf16, fp32, 4, 1, 8, 16, 4);

// REGISTER_BWD_LAUNCHER(25600, fp32, fp32, fp32, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(25600, fp16, fp16, fp16, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(25600, fp16, fp32, fp16, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(25600, bf16, bf16, bf16, fp32, 5, 1, 4, 16, 4);
// REGISTER_BWD_LAUNCHER(25600, bf16, fp32, bf16, fp32, 5, 1, 4, 16, 4);

// REGISTER_BWD_LAUNCHER(30720, fp32, fp32, fp32, fp32, 4, 1, 8, 8, 4);
// REGISTER_BWD_LAUNCHER(30720, fp16, fp16, fp16, fp32, 4, 1, 8, 4, 4);
// REGISTER_BWD_LAUNCHER(30720, fp16, fp32, fp16, fp32, 4, 1, 8, 8, 4);
// REGISTER_BWD_LAUNCHER(30720, bf16, bf16, bf16, fp32, 4, 1, 8, 4, 4);
// REGISTER_BWD_LAUNCHER(30720, bf16, fp32, bf16, fp32, 4, 1, 8, 8, 4);

// REGISTER_BWD_LAUNCHER(32768, fp32, fp32, fp32, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(32768, fp16, fp16, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(32768, fp16, fp32, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(32768, bf16, bf16, bf16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(32768, bf16, fp32, bf16, fp32, 4, 1, 8, 16, 4);

// REGISTER_BWD_LAUNCHER(40960, fp32, fp32, fp32, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(40960, fp16, fp16, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(40960, fp16, fp32, fp16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(40960, bf16, bf16, bf16, fp32, 4, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(40960, bf16, fp32, bf16, fp32, 4, 1, 8, 16, 4);

// REGISTER_BWD_LAUNCHER(49152, fp32, fp32, fp32, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(49152, fp16, fp16, fp16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(49152, fp16, fp32, fp16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(49152, bf16, bf16, bf16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(49152, bf16, fp32, bf16, fp32, 8, 1, 8, 16, 4);

// REGISTER_BWD_LAUNCHER(65536, fp32, fp32, fp32, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(65536, fp16, fp16, fp16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(65536, fp16, fp32, fp16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(65536, bf16, bf16, bf16, fp32, 8, 1, 8, 16, 4);
// REGISTER_BWD_LAUNCHER(65536, bf16, fp32, bf16, fp32, 8, 1, 8, 16, 4);
