#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

#include <torch/python.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

void apply_rotary_cuda(const torch::Tensor x1, const torch::Tensor x2,
                       const torch::Tensor cos, const torch::Tensor sin,
                       torch::Tensor out1, torch::Tensor out2,
                       const bool conj) {
    auto iter = at::TensorIteratorConfig()
        .add_output(out1)
        .add_output(out2)
        .add_input(x1)
        .add_input(x2)
        .add_input(cos)
        .add_input(sin)
        .check_all_same_dtype(false)
        .promote_inputs_to_common_dtype(false)
        .build();

    if (!conj) {
        AT_DISPATCH_FLOATING_TYPES_AND2(at::kBFloat16, at::kHalf, x1.scalar_type(), "rotary_kernel", [&] {
            at::native::gpu_kernel_multiple_outputs(
                iter, [] GPU_LAMBDA (scalar_t x1, scalar_t x2, scalar_t cos,
                                    scalar_t sin) -> thrust::tuple<scalar_t, scalar_t> {
                scalar_t out1 = float(x1) * float(cos) - float(x2) * float(sin);
                scalar_t out2 = float(x1) * float(sin) + float(x2) * float(cos);
                return {out1, out2};
            });
        });
    } else {
        AT_DISPATCH_FLOATING_TYPES_AND2(at::kBFloat16, at::kHalf, x1.scalar_type(), "rotary_kernel", [&] {
            at::native::gpu_kernel_multiple_outputs(
                iter, [] GPU_LAMBDA (scalar_t x1, scalar_t x2, scalar_t cos,
                                    scalar_t sin) -> thrust::tuple<scalar_t, scalar_t> {
                scalar_t out1 = float(x1) * float(cos) + float(x2) * float(sin);
                scalar_t out2 = -float(x1) * float(sin) + float(x2) * float(cos);
                return {out1, out2};
            });
        });
    }
}