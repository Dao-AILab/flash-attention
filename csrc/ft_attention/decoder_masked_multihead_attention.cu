#include "hip/hip_runtime.h"
// Adapted from from FasterTransformer v5.2.1
// https://github.com/NVIDIA/FasterTransformer/blob/release/v5.2.1_tag/src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_128.cu
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "decoder_masked_multihead_attention.h"
#include "decoder_masked_multihead_attention_utils.h"
#include "cuda_bf16_wrapper.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

#include "decoder_masked_multihead_attention_template.hpp"

////////////////////////////////////////////////////////////////////////////////////////////////////

#define MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, THDS_PER_KEY, THDS_PER_VALUE, THDS_PER_BLOCK, DO_CROSS_ATTENTION, stream)    \
    size_t smem_sz = mmha::smem_size_in_bytes<T, DO_CROSS_ATTENTION>(params, THDS_PER_VALUE, THDS_PER_BLOCK);          \
    auto kernel = mmha::masked_multihead_attention_kernel<T, Dh, Dh_MAX, THDS_PER_KEY, THDS_PER_VALUE,                 \
                                                          THDS_PER_BLOCK, DO_CROSS_ATTENTION>;                         \
    if (smem_sz >= 48 * 1024) {                                                                                        \
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_sz);                            \
    }                                                                                                                  \
    dim3 grid(params.nnz_head_idx == nullptr ? params.num_heads : params.nnz_heads, params.batch_size);                \
    kernel<<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(params)

////////////////////////////////////////////////////////////////////////////////////////////////////

// !!! Specialize the launcher for Cross attention
template<typename T, int Dh, int Dh_MAX, typename KERNEL_PARAMS_TYPE>
void mmha_launch_kernel(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    constexpr int  THREADS_PER_VALUE  = Dh_MAX * sizeof(T) / 16;
    constexpr bool DO_CROSS_ATTENTION = std::is_same<KERNEL_PARAMS_TYPE, Cross_multihead_attention_params<T>>::value;
    int            tlength            = (DO_CROSS_ATTENTION) ? params.memory_max_len : params.timestep;
    // printf("tlength, CROSS_ATTENTION = %d, %d\n", tlength, DO_CROSS_ATTENTION);
    if (tlength < 32) {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, DO_CROSS_ATTENTION, stream);
    }
    else if (tlength < 2048) {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 2, THREADS_PER_VALUE, 128, DO_CROSS_ATTENTION, stream);
    }
    else {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 1, THREADS_PER_VALUE, 256, DO_CROSS_ATTENTION, stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#undef MMHA_LAUNCH_KERNEL

template<typename T, typename KERNEL_PARAMS_TYPE>
void multihead_attention_(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    switch (params.hidden_size_per_head) {
        case 32:
            mmha_launch_kernel<T, 32, 32, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 48:
            mmha_launch_kernel<T, 48, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 64:
            mmha_launch_kernel<T, 64, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 80:
            mmha_launch_kernel<T, 80, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 96:
            mmha_launch_kernel<T, 96, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 128:
            mmha_launch_kernel<T, 128, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 160:
            mmha_launch_kernel<T, 160, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 192:
            mmha_launch_kernel<T, 192, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 224:
            mmha_launch_kernel<T, 224, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 256:
            mmha_launch_kernel<T, 256, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        default:
            assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<float>& params, const hipStream_t& stream)
{
    multihead_attention_<float, Masked_multihead_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    multihead_attention_<uint16_t, Masked_multihead_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void masked_multihead_attention(const Masked_multihead_attention_params<__hip_bfloat16>& params,
                                const hipStream_t&                                     stream)
{
    multihead_attention_<__hip_bfloat16, Masked_multihead_attention_params<__hip_bfloat16>>(params, stream);
}
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////

void cross_multihead_attention(const Cross_multihead_attention_params<float>& params, const hipStream_t& stream)
{
    multihead_attention_<float, Cross_multihead_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void cross_multihead_attention(const Cross_multihead_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    multihead_attention_<uint16_t, Cross_multihead_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void cross_multihead_attention(const Cross_multihead_attention_params<__hip_bfloat16>& params,
                               const hipStream_t&                                    stream)
{
    multihead_attention_<__hip_bfloat16, Cross_multihead_attention_params<__hip_bfloat16>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////
