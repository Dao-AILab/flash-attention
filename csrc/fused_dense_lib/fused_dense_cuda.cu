// Adapted from https://github.com/NVIDIA/apex/blob/master/csrc/fused_dense_cuda.cu
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11000
// includes cublaslt
#include <hipblaslt.h>
#endif

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    at::Half* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

// BF16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    at::BFloat16* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_16BF,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    int heuristic,
    const void* gelu_in,
    const void* bias) {
  bool save_gelu_in = gelu_in != nullptr;
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (save_gelu_in) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));
  }

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS : HIPBLASLT_EPILOGUE_GELU_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          // &heuristicResult.algo,
                          // TD [2022-04-29] Somehow algo 0 and 2 are a lot slower than other algos
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    int heuristic,
    const void* gelu_in,
    const void* bias) {
  bool save_gelu_in = gelu_in != nullptr;
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX : HIPBLASLT_EPILOGUE_GELU;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (save_gelu_in) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));
  }

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
    epilogue = save_gelu_in ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS : HIPBLASLT_EPILOGUE_GELU_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          // &heuristicResult.algo,
                          // TD [2022-04-29] Somehow algo 0 and 2 are a lot slower than other algos
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float *beta, /* host pointer */
    at::BFloat16* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    int heuristic,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  constexpr int requestedAlgoCount = 5;
  hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = {0};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16BF, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16BF, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16BF, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, requestedAlgoCount, heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          &heuristicResult[heuristic].algo,
                          // NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

#endif

template <typename T>
int linear_bias_forward_cuda(at::Tensor input, T *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    const float beta_one       = 1.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    input.data_ptr<T>(),
    in_features,
    &beta_zero, /* host pointer */
    output.data_ptr<T>(),
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(bias.data_ptr<T>()));
#endif
    if (status != 0){
        output.copy_(bias);
        status = gemm_bias(
          handle,
          HIPBLAS_OP_T,
          HIPBLAS_OP_N,
          out_features,
          batch_size,
          in_features,
          &alpha,
          weight,
          in_features,
          input.data_ptr<T>(),
          in_features,
          &beta_one,
          output.data_ptr<T>(),
          out_features);
    }
    return status;
}

    
template <typename T>
int linear_bias_backward_cuda(T *input, T *weight, T *d_output, int in_features, int batch_size, int out_features, T *d_weight, T *d_bias, T *d_input, bool residual, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero      = 0.0;
    const float beta           = residual ? 1.0 : 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    in_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    input,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_weight,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(d_bias));
#endif
    

    if (status != 0){
    
        status = gemm_bias(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          in_features,
          out_features,
          batch_size,
          &alpha,
          input,
          in_features,
          d_output,
          out_features,
          &beta_zero,
          d_weight,
          in_features);
    }
    
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      in_features,
      batch_size,
      out_features,
      &alpha,
      weight,
      in_features,
      d_output,
      out_features,
      &beta,
      d_input,
      in_features);
    return status;

}

template <typename T>
int linear_bias_wgrad_cuda(T *input, T *d_output, int in_features, int batch_size, int out_features, T *d_weight, T *d_bias, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero      = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    in_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    input,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_weight,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(d_bias));
#endif


    if (status != 0){

        status = gemm_bias(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          in_features,
          out_features,
          batch_size,
          &alpha,
          input,
          in_features,
          d_output,
          out_features,
          &beta_zero,
          d_weight,
          in_features);
    }

    return status;
}

template <typename T>
int linear_gelu_forward_cuda(T *input, T *weight, T *bias, int in_features, int batch_size, int out_features, int heuristic, T *output, T *gelu_in, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_gelu_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    input,
    in_features,
    &beta_zero, /* host pointer */
    output,
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    heuristic,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(bias));
    return status;
#else
    return 1;
#endif
}

template <typename T>
int linear_gelu_linear_backward_cuda(T *input, T *gelu_in, T *output1, T *weight1, T *weight2, T *d_output1, T *d_output2, int in_features, int batch_size, int hidden_features, int out_features, int heuristic, T *d_weight1, T *d_weight2, T *d_bias1, T *d_bias2, T *d_input, bool residual, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero      = 0.0;
    const float beta           = residual ? 1.0 : 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
//wgrad for first gemm
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    hidden_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    output1,
    hidden_features,
    d_output2,
    out_features,
    &beta_zero, /* host pointer */
    d_weight2,
    hidden_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(d_bias2));
//dgrad for second GEMM
    status = gemm_dgelu_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    hidden_features,
    batch_size,
    out_features,
    &alpha, /* host pointer */
    weight2,
    hidden_features,
    d_output2,
    out_features,
    &beta_zero, /* host pointer */
    d_output1,
    hidden_features,
    lt_workspace,
    1 << 22,
    stream,
    heuristic,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(d_bias1));
//wgrad for the first GEMM
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      in_features,
      hidden_features,
      batch_size,
      &alpha,
      input,
      in_features,
      d_output1,
      hidden_features,
      &beta_zero,
      d_weight1,
      in_features);

//dgrad for the first GEMM
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      in_features,
      batch_size,
      hidden_features,
      &alpha,
      weight1,
      in_features,
      d_output1,
      hidden_features,
      &beta,
      d_input,
      in_features);
#endif
    return status;

}


template int linear_bias_forward_cuda<at::Half>(at::Tensor input, at::Half *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);
template int linear_bias_forward_cuda<at::BFloat16>(at::Tensor input, at::BFloat16 *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_backward_cuda<at::Half>(at::Half *input, at::Half *weight, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, at::Half *d_input, bool residual, void *lt_workspace) ;
template int linear_bias_backward_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *weight, at::BFloat16 *d_output, int in_features, int batch_size, int out_features, at::BFloat16 *d_weight, at::BFloat16 *d_bias, at::BFloat16 *d_input, bool residual, void *lt_workspace) ;

template int linear_bias_wgrad_cuda<at::Half>(at::Half *input, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, void *lt_workspace) ;
template int linear_bias_wgrad_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, int in_features, int batch_size, int out_features, at::BFloat16 *d_weight, at::BFloat16 *d_bias, void *lt_workspace) ;

template int linear_gelu_forward_cuda<at::Half>(at::Half *input, at::Half *weight, at::Half *bias, int in_features, int batch_size, int out_features, int heuristic, at::Half *output, at::Half *gelu_in, void *lt_workspace) ;
template int linear_gelu_forward_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *weight, at::BFloat16 *bias, int in_features, int batch_size, int out_features, int heuristic, at::BFloat16 *output, at::BFloat16 *gelu_in, void *lt_workspace) ;

template int linear_gelu_linear_backward_cuda<at::Half>(at::Half *input, at::Half *gelu_in, at::Half *output1, at::Half *weight1, at::Half *weight2, at::Half *d_output1, at::Half *d_output2, int in_features, int batch_size, int hidden_features, int out_features, int heuristic, at::Half *d_weight1, at::Half *d_weight2, at::Half *d_bias1, at::Half *d_bias2, at::Half *d_input, bool residual, void *lt_workspace);
template int linear_gelu_linear_backward_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *gelu_in, at::BFloat16 *output1, at::BFloat16 *weight1, at::BFloat16 *weight2, at::BFloat16 *d_output1, at::BFloat16 *d_output2, int in_features, int batch_size, int hidden_features, int out_features, int heuristic, at::BFloat16 *d_weight1, at::BFloat16 *d_weight2, at::BFloat16 *d_bias1, at::BFloat16 *d_bias2, at::BFloat16 *d_input, bool residual, void *lt_workspace);
