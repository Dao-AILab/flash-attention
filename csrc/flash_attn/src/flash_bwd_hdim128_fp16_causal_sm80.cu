// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

template<>
void run_mha_bwd_<cutlass::half_t, 128, true>(Flash_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_hdim128<cutlass::half_t, true>(params, stream);
}
