#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "static_switch.h"
#include "fp16_switch.h"
#include "fmha.h"
#include "fmha_fprop_kernel_1xN.h"

template<typename Kernel_traits, bool Is_dropout, bool Is_causal, bool Return_softmax, bool Need_attn_mask, bool Need_attn_bias>
__global__ void fmha_fprop_fp16_sm80_loop_kernel(FMHA_fprop_params params) {
    fmha::device_1xN_loop<Kernel_traits, Is_dropout, Is_causal, Return_softmax, Need_attn_mask, Need_attn_bias>(params);
}

template<typename Kernel_traits>
void run_fmha_fp16_sm80_loop_(Launch_params<FMHA_fprop_params> &launch_params,
                              const bool configure) {
    constexpr int blocksize_c = Kernel_traits::Cta_tile_p::N;
    const int loop_steps = (launch_params.params.seqlen_k + blocksize_c - 1) / blocksize_c;

    if (configure) {
        using Mma_tile_p = fmha::Hmma_tile<typename Kernel_traits::Cta_tile_p>;
        constexpr int M = Kernel_traits::Cta_tile_p::M;
        size_t STEPS = (launch_params.params.seqlen_q + M - 1) / M;
        constexpr size_t MMAS_M = Mma_tile_p::MMAS_M;
        constexpr size_t MMAS_N = Mma_tile_p::MMAS_N;
        size_t elts_per_head = STEPS * MMAS_M * MMAS_N * 8 * loop_steps;
        launch_params.elts_per_thread = elts_per_head;
        return;
    }

    constexpr int smem_size_softmax_lse = Kernel_traits::Smem_dp_sum::BYTES_PER_TILE;
    // Don't need smem_size_softmax_lse if we're not looping
    const int smem_size = fmha::get_dynamic_smem_size<Kernel_traits>()
        + (loop_steps > 1 ? smem_size_softmax_lse : 0);

    bool has_attn_mask = !(launch_params.params.attn_mask_ptr == nullptr);
    bool has_attn_bias = !(launch_params.params.attn_bias_ptr == nullptr);

    if (has_attn_mask) 
    {
        if (has_attn_bias) {
            // Work-around for gcc 7. It doesn't like nested BOOL_SWITCH.
            // https://github.com/kokkos/kokkos-kernels/issues/349
            // https://github.com/HazyResearch/flash-attention/issues/21
            BOOL_SWITCH(launch_params.is_dropout, IsDropoutConst, [&] {
                auto kernel = launch_params.params.is_causal
                    ? (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, true, true, true>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, false, true, true>)
                    : (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, true, true, true>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, false, true, true>);
                if( smem_size >= 48 * 1024 ) {
                    FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
                }
                dim3 grid(launch_params.params.b, launch_params.params.h);

                // printf("grid size: %d %d\n", launch_params.params.b, launch_params.params.h);
                // printf("block size: %d\n", Kernel_traits::THREADS);
                kernel<<<grid, Kernel_traits::THREADS, smem_size, launch_params.stream>>>(
                    launch_params.params);
                FMHA_CHECK_CUDA(hipPeekAtLastError());
            });
        }else{
            // Work-around for gcc 7. It doesn't like nested BOOL_SWITCH.
            // https://github.com/kokkos/kokkos-kernels/issues/349
            // https://github.com/HazyResearch/flash-attention/issues/21
            BOOL_SWITCH(launch_params.is_dropout, IsDropoutConst, [&] {
                auto kernel = launch_params.params.is_causal
                    ? (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, true, true, false>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, false, true, false>)
                    : (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, true, true, false>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, false, true, false>);
                if( smem_size >= 48 * 1024 ) {
                    FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
                }
                dim3 grid(launch_params.params.b, launch_params.params.h);

                // printf("grid size: %d %d\n", launch_params.params.b, launch_params.params.h);
                // printf("block size: %d\n", Kernel_traits::THREADS);
                kernel<<<grid, Kernel_traits::THREADS, smem_size, launch_params.stream>>>(
                    launch_params.params);
                FMHA_CHECK_CUDA(hipPeekAtLastError());
            });
        }
    }else{
        if (has_attn_bias) {
            // Work-around for gcc 7. It doesn't like nested BOOL_SWITCH.
            // https://github.com/kokkos/kokkos-kernels/issues/349
            // https://github.com/HazyResearch/flash-attention/issues/21
            BOOL_SWITCH(launch_params.is_dropout, IsDropoutConst, [&] {
                auto kernel = launch_params.params.is_causal
                    ? (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, true, false, true>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, false, false, true>)
                    : (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, true, false, true>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, false, false, true>);
                if( smem_size >= 48 * 1024 ) {
                    FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
                }
                dim3 grid(launch_params.params.b, launch_params.params.h);

                // printf("grid size: %d %d\n", launch_params.params.b, launch_params.params.h);
                // printf("block size: %d\n", Kernel_traits::THREADS);
                kernel<<<grid, Kernel_traits::THREADS, smem_size, launch_params.stream>>>(
                    launch_params.params);
                FMHA_CHECK_CUDA(hipPeekAtLastError());
            });
        }else{
            // Work-around for gcc 7. It doesn't like nested BOOL_SWITCH.
            // https://github.com/kokkos/kokkos-kernels/issues/349
            // https://github.com/HazyResearch/flash-attention/issues/21
            BOOL_SWITCH(launch_params.is_dropout, IsDropoutConst, [&] {
                auto kernel = launch_params.params.is_causal
                    ? (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, true, false, false>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, true, false, false, false>)
                    : (launch_params.return_softmax
                    ? &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, true, false, false>
                    : &fmha_fprop_fp16_sm80_loop_kernel<Kernel_traits, IsDropoutConst, false, false, false, false>);
                if( smem_size >= 48 * 1024 ) {
                    FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                        kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
                }
                dim3 grid(launch_params.params.b, launch_params.params.h);

                // printf("grid size: %d %d\n", launch_params.params.b, launch_params.params.h);
                // printf("block size: %d\n", Kernel_traits::THREADS);
                kernel<<<grid, Kernel_traits::THREADS, smem_size, launch_params.stream>>>(
                    launch_params.params);
                FMHA_CHECK_CUDA(hipPeekAtLastError());
            });
        }
    }
}

void run_fmha_fp16_sm80(Launch_params<FMHA_fprop_params> &launch_params,
                        const bool configure) {
    FP16_SWITCH(launch_params.params.is_bf16, [&] {
        auto dprops = at::cuda::getCurrentDeviceProperties();
        if (launch_params.params.d == 16) {
            if( launch_params.params.seqlen_k == 128 ) {
                using Kernel_traits = FMHA_kernel_traits<128, 16, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } 
            else if( launch_params.params.seqlen_k == 256 ) {
                using Kernel_traits = FMHA_kernel_traits<256, 16, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } else {
                // TD [2022-05-15] 512 gives wrong results rn
                // using Kernel_traits = FMHA_kernel_traits<512, 16, 16, 1, 4, 0x08u, elem_type>;
                using Kernel_traits = FMHA_kernel_traits<256, 16, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            }
        }
        else if (launch_params.params.d == 32) {
            if( launch_params.params.seqlen_k == 128 ) {
                using Kernel_traits = FMHA_kernel_traits<128, 32, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } else if( launch_params.params.seqlen_k == 256 ) {
                using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } else {
                using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            }
        } 
        else if (launch_params.params.d == 64) {
            if( launch_params.params.seqlen_k == 128 ) {
                using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } else if( launch_params.params.seqlen_k >= 256 ) {
                if (dprops->major == 8 && dprops->minor >= 0) {
                    using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 4, 0x08u, elem_type>;
                    run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
                } else if (dprops->major == 7 && dprops->minor == 5) {
                    if (launch_params.is_dropout) { // Need to use the same block size as backward
                        using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
                        run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
                    } else {
                        using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 4, 0x08u, elem_type>;
                        run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
                    }
                }
            }
        } else if (launch_params.params.d == 128) {
            if( launch_params.params.seqlen_k == 128 ) {
                using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 4, 0x08u, elem_type>;
                run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
            } else {
                if (dprops->major == 8 && dprops->minor == 0 && !launch_params.is_dropout) {
                    // TD [2022-06-05] Keep K in registers to reduce register spilling
                    // Gives about 6% speedup compared to using block size 128.
                    using Kernel_traits = FMHA_kernel_traits<256, 128, 16, 1, 4, 0x18u, elem_type>;
                    run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
                } else {  // Need to use the same block size as backward
                    using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 4, 0x08u, elem_type>;
                    run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
                }
            }
        }
        // if (launch_params.params.d == 64) {
        //     // using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
        //     // using Kernel_traits = FMHA_kernel_traits<64, 64, 16, 1, 4, 0x08u, elem_type>;
        //     // using Kernel_traits = FMHA_kernel_traits<512, 64, 16, 1, 8, 0x08u, elem_type>;
        //     using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
        //     run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        // }
        // if (launch_params.params.d == 64) {
        //     if( launch_params.params.seqlen_k == 128 ) {
        //         using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
        //         run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //     } else if( launch_params.params.seqlen_k >= 256 ) {
        //         if (dprops->major == 8 && dprops->minor >= 0) {
        //             using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 4, 0x08u, elem_type>;
        //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //         } else if (dprops->major == 7 && dprops->minor == 5) {
        //             if (launch_params.is_dropout) { // Need to use the same block size as backward
        //                 using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
        //                 run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //             } else {
        //                 using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 4, 0x08u, elem_type>;
        //                 run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //             }
        //         }
        //     }
        // }
        // if (launch_params.params.d == 128) {
        //     if( launch_params.params.seqlen_k == 128 ) {
        //         using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 4, 0x08u, elem_type>;
        //         run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //     } else {
        //         if (dprops->major == 8 && dprops->minor >= 0 && !launch_params.is_dropout) {
        //             // TD [2022-06-05] Keep K in registers to reduce register spilling
        //             // Gives about 6% speedup compared to using block size 128.
        //             using Kernel_traits = FMHA_kernel_traits<256, 128, 16, 1, 4, 0x18u, elem_type>;
        //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //         } else {  // Need to use the same block size as backward
        //             using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 4, 0x08u, elem_type>;
        //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params, configure);
        //         }
        //     }
        // }
    });
}