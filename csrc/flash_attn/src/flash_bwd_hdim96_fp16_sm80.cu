// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_bwd_launch_template.h"

// template<>
// void run_mha_bwd_<cutlass::half_t, 96>(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
//     using elem_type = cutlass::half_t;
//     if (params.h == params.h_k) {
//         // run_flash_bwd<Flash_bwd_kernel_traits<96, 64, 128, 8, 2, 4, 4, true, false, elem_type>>(params, stream, configure);
//         // This is very slightly faster
//         run_flash_bwd<Flash_bwd_kernel_traits<96, 64, 128, 8, 2, 4, 4, false, false, elem_type>>(params, stream, configure);
//     } else {
//         run_flash_bwd_seqq_parallel<Flash_bwd_kernel_traits<96, 128, 64, 8, 4, 4, 4, false, false, elem_type>>(params, stream, configure);
//     }
// }

template<>
void run_mha_bwd_<cutlass::half_t, 96>(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    run_mha_bwd_hdim96<cutlass::half_t>(params, stream, configure);
}