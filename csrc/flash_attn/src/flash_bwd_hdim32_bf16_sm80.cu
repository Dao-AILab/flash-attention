// Copyright (c) 2023, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

template<>
void run_mha_bwd_<cutlass::bfloat16_t, 32>(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    run_mha_bwd_hdim32<cutlass::bfloat16_t>(params, stream, configure);
}
