// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_qkdim128_vdim256_sm80.h"

template<>
void run_mha_bwd_<cutlass::bfloat16_t, 128, 256, false>(Flash_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_qkdim128_vdim256<cutlass::bfloat16_t, false>(params, stream);
}
