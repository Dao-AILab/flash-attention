// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_fwd_qkdim96_vdim192_sm80.h"

template<>
void run_mha_fwd_<cutlass::bfloat16_t, 96, 192, true>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_qkdim96_vdim192<cutlass::bfloat16_t, true>(params, stream);
}
