// Copyright (c) 2023, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

template<>
void run_mha_bwd_<cutlass::half_t, 64>(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    run_mha_bwd_hdim64<cutlass::half_t>(params, stream, configure);
}
