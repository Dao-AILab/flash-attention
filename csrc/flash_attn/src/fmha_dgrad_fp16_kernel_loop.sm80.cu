#include "hip/hip_runtime.h"
/* Copyright (c) 2022, Tri Dao.
 */

#include "static_switch.h"
#include "fp16_switch.h"
#include "fmha.h"
#include "fmha_dgrad_kernel_1xN_loop.h"

// Pick whether we should parallelize across seqlen_k (num_splits > 1) or not (num_splits=1).
// Parallelizing will have better occupancy, but has some overhead due to having to zero out
// dq_tmp and having to copy dq_tmp to dq.
int num_splits_heuristic_bwd(int batch_nheads, int num_SMs, int ctas_per_sm, int seqlen,
                             int blocksize, bool is_causal) {
    float n_waves_1 = float(batch_nheads) / (num_SMs * ctas_per_sm);
    float eff_1 = n_waves_1 / ceil(n_waves_1);
    int num_splits_parallel = seqlen / blocksize;
    float n_waves_parallel = float(batch_nheads * num_splits_parallel) / (num_SMs * ctas_per_sm);
    float eff_parallel_raw = n_waves_parallel / ceil(n_waves_parallel);
    float discount_factor;
    if (!is_causal) {
        discount_factor = 1.f + float(blocksize) / seqlen;
    } else {  // For causal, parallelizing seems to help with load-balancing as well
        // For example, if headdim=128, seqlen >= 1280 always prefers parallel
        if (seqlen / blocksize >= 10) return num_splits_parallel;
        discount_factor = 1.f + 0.5 * float(blocksize) / seqlen;
    }
    float eff_parallel = eff_parallel_raw / discount_factor;
    return eff_1 >= eff_parallel ? 1 : num_splits_parallel;
}

template<typename Kernel_traits>
__global__ void fmha_dgrad_dot_do_o_kernel(FMHA_dgrad_params params) {
    fmha::compute_dot_do_o<Kernel_traits>(params);
}

template<typename Kernel_traits, bool Is_dropout, bool Is_causal, int loop_steps=-1>
__global__ void fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel(FMHA_dgrad_params params) {
    fmha::compute_dq_dk_dv_1xN<Kernel_traits, Is_dropout, Is_causal, loop_steps>(params);
}

template<typename Kernel_traits, bool Is_dropout, bool Is_causal>
__global__ void fmha_dgrad_fp16_sm80_dq_dk_dv_loop_seqparallel_kernel(FMHA_dgrad_params params) {
    fmha::compute_dq_dk_dv_seqparallel<Kernel_traits, Is_dropout, Is_causal>(params);
}

template<typename Kernel_traits>
void run_fmha_dgrad_fp16_sm80_loop_(FMHA_dgrad_params &params, hipStream_t stream, const bool configure) {
    constexpr int smem_size_softmax = Kernel_traits::Cta_tile_p::M * Kernel_traits::Cta_tile_p::WARPS_N * sizeof(float);
    constexpr int smem_size_q = Kernel_traits::Smem_tile_q::BYTES_PER_TILE;
    constexpr int smem_size_v = Kernel_traits::Smem_tile_v::BYTES_PER_TILE;
    constexpr int smem_size_dq = Kernel_traits::Smem_tile_o::BYTES_PER_TILE;

    using Smem_tile_s = fmha::Smem_tile_mma_transposed<typename Kernel_traits::Cta_tile_p>;
    constexpr int smem_size_s = Smem_tile_s::BYTES_PER_TILE;
    static_assert(smem_size_s == 16 * Kernel_traits::Cta_tile_p::N * 2);
    static_assert(smem_size_dq == 16 * Kernel_traits::Cta_tile_p::K * 4 * Kernel_traits::Cta_tile_p::WARPS_N);

    constexpr int smem_size_dq_dk_dv = smem_size_q * 2 + smem_size_v * (Kernel_traits::V_IN_REGS ? 1 : 2) + smem_size_dq + smem_size_s * 2;
    constexpr int blocksize_c = Kernel_traits::Cta_tile_p::N;
    // printf("blocksize_c = %d, WARPS_N = %d, Smem size = %d\n", blocksize_c, Kernel_traits::Cta_tile_p::WARPS_N, smem_size_dq_dk_dv);

    bool is_dropout = params.p_dropout < 1.f;  // params.p_dropout is the probability of "keeping"
    // Work-around for gcc 7. It doesn't like nested BOOL_SWITCH.
    BOOL_SWITCH(is_dropout, IsDropoutConst, [&] {
        auto kernel = params.is_causal
            ? &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, true>
            : &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, false>;
        if (params.seqlen_k == blocksize_c) {
            kernel = params.is_causal
                ? &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, true, /*loop_steps=*/1>
                : &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, false, /*loop_steps=*/1>;
        } else if (params.seqlen_k == blocksize_c * 2) {
            kernel = params.is_causal
                ? &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, true, /*loop_steps=*/2>
                : &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_kernel<Kernel_traits, IsDropoutConst, false, /*loop_steps=*/2>;
        }
        auto kernel_seqparallel = params.is_causal
            ? &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_seqparallel_kernel<Kernel_traits, IsDropoutConst, true>
            : &fmha_dgrad_fp16_sm80_dq_dk_dv_loop_seqparallel_kernel<Kernel_traits, IsDropoutConst, false>;
        if( smem_size_dq_dk_dv >= 48 * 1024 ) {
            FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size_dq_dk_dv));
            FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel_seqparallel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size_dq_dk_dv));
        }
        // Automatically set num_splits to maximize occupancy
        if (params.num_splits <= 0) {
            int ctas_per_sm;
            hipError_t status_ = hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &ctas_per_sm, kernel, Kernel_traits::THREADS, smem_size_dq_dk_dv);
            auto dprops = at::cuda::getCurrentDeviceProperties();
            // printf("CTAS_PER_SM = %d, nSMs = %d\n", ctas_per_sm, dprops->multiProcessorCount);
            constexpr int M = Kernel_traits::Cta_tile_p::M;
            // We don't want more than 10 splits due to numerical error.
            // Numerical error on dk/dv scales as sqrt(num_splits).
            params.num_splits = num_splits_heuristic_bwd(
                params.b * params.h, dprops->multiProcessorCount,
                ctas_per_sm, params.seqlen_k, blocksize_c, params.is_causal
            );
        }
        if (configure) return;
        if (params.num_splits == 1) {
            dim3 grid(params.b, params.h, params.num_splits);
            kernel<<<grid, Kernel_traits::THREADS, smem_size_dq_dk_dv, stream>>>(params);
        } else {
            dim3 grid_dot(params.b, params.h, (params.seqlen_q + 128 - 1) / 128);
            fmha_dgrad_dot_do_o_kernel<Kernel_traits><<<grid_dot, Kernel_traits::THREADS, 0, stream>>>(params);
            int num_splits = params.seqlen_k / blocksize_c;  // seqlen_k is divisible by blocksize_c
            dim3 grid(params.b, params.h, num_splits);
            kernel_seqparallel<<<grid, Kernel_traits::THREADS, smem_size_dq_dk_dv, stream>>>(params);
        }
        FMHA_CHECK_CUDA(hipPeekAtLastError());
    });
}

void run_fmha_dgrad_fp16_sm80(FMHA_dgrad_params &params, hipStream_t stream, const bool configure) {
    // work around for MSVC issue
    FP16_SWITCH(params.is_bf16, [&] {
        auto dprops = at::cuda::getCurrentDeviceProperties();
        if (params.d <= 32) {
            if (params.seqlen_k == 128) {
                using Kernel_traits = FMHA_kernel_traits<128, 32, 16, 1, 8, 0x08u, elem_type>;
                run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
            } else if (params.seqlen_k >= 256) {
                using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 8, 0x08u, elem_type>;
                run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
            }
        } else if (params.d <= 64) {
            if (params.seqlen_k == 128) {
                using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 8, 0x08u, elem_type>;
                run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
            } else if (params.seqlen_k >= 256) {
                if (dprops->major == 8 && dprops->minor == 0) {
                    // Don't share smem for K & V, and don't keep V in registers
                    // This speeds things up by 2-3% by avoiding register spills, but it
                    // uses more shared memory, which is fine on A100 but not other GPUs.
                    // For other GPUs, we keep V in registers.
                    using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 8, 0x100u, elem_type>;
                    run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
                } else if (dprops->major == 8 && dprops->minor > 0) {
                    using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 8, 0x08u, elem_type>;
                    run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
                } else if (dprops->major == 7 && dprops->minor == 5) {
                    using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 8, 0x08u, elem_type>;
                    run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
                }
            }
        } else if (params.d <= 128) {
            using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 8, 0x100u, elem_type>;
            run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream, configure);
        }
    });
}