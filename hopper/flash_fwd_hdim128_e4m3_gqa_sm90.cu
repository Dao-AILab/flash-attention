// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_gqa_<cutlass::float_e4m3_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128_fp8_gqa_decoding<cutlass::float_e4m3_t>(params, stream);
}
