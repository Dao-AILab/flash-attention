// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_combine_launch_template.h"

template void run_mha_fwd_combine_<float, float, 64>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<float, float, 128>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<float, float, 256>(Flash_fwd_params &params, hipStream_t stream);

template void run_mha_fwd_combine_<cutlass::half_t, float, 64>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<cutlass::half_t, float, 128>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<cutlass::half_t, float, 256>(Flash_fwd_params &params, hipStream_t stream);

template void run_mha_fwd_combine_<cutlass::bfloat16_t, float, 64>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<cutlass::bfloat16_t, float, 128>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_combine_<cutlass::bfloat16_t, float, 256>(Flash_fwd_params &params, hipStream_t stream);
