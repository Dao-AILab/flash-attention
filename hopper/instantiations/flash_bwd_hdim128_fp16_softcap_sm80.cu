// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different template instantiations to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

#ifndef FLASHATTENTION_DISABLE_SM8x
#ifndef FLASHATTENTION_DISABLE_HDIM128
template<>
void run_mha_bwd_<80, cutlass::half_t, 128, true>(Flash_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_hdim128<80, cutlass::half_t, true>(params, stream);
}
template<>
void run_mha_bwd_<86, cutlass::half_t, 128, true>(Flash_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_hdim128<86, cutlass::half_t, true>(params, stream);
}
#endif
#endif
