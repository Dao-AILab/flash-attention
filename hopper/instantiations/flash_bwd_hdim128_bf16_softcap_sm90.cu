// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different template instantiations to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_bwd_launch_template.h"

#ifndef FLASHATTENTION_DISABLE_HDIM128
template<>
void run_mha_bwd_<90, cutlass::bfloat16_t, 128, true>(Flash_bwd_params &params, hipStream_t stream) {
    run_mha_bwd_hdim128<90, cutlass::bfloat16_t, true>(params, stream);
}
#endif
